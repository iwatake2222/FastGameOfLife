#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include <string.h>
#include "algorithmCudaNormal.h"

namespace AlgorithmCudaNormal
{
#if 0
}	// indent guard
#endif


__global__ void loop(int* matDst, int *matSrc, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	//if (x >= width || y >= height) {
	//	printf("%d %d\n", x, y);
	//	return;
	//}

	int cnt = 0;
	for (int yy = y - 1; yy <= y + 1; yy++) {
		int roundY = yy;
		if (roundY >= height) roundY = 0;
		if (roundY < 0) roundY = height - 1;
		for (int xx = x - 1; xx <= x + 1; xx++) {
			int roundX = xx;
			if (roundX >= width) roundX = 0;
			if (roundX < 0) roundX = width - 1;
			if (matSrc[width * roundY + roundX] != 0) {
				cnt++;
			}
		}
	}

	int yLine = y * width;
	if (matSrc[yLine + x] == 0) {
		if (cnt == 3) {
			// birth
			matDst[yLine + x] = 1;
		} else {
			// keep dead
			matDst[yLine + x] = 0;
		}
	} else {
		if (cnt <= 2 || cnt >= 5) {
			// die
			matDst[yLine + x] = 0;
		} else {
			// keep alive (age++)
			matDst[yLine + x] = matSrc[yLine + x] + 1;
		}
	}
}

#if 0
/* always copy from host to device */
void logicForOneGeneration(ALGORITHM_CUDA_NORMAL_PARAM *param, int* matDst, int* matSrc, int width, int height)
{
	hipMemcpy(param->devMatSrc, matSrc, width * height * sizeof(int), hipMemcpyHostToDevice);

	int blocksizeW = 16;
	int blocksizeH = 16;
	dim3 block(blocksizeW, blocksizeH);
	dim3 grid(width / blocksizeW, height / blocksizeH);
	loop <<<grid, block >>> (param->devMatDst, param->devMatSrc, width, height);
	hipDeviceSynchronize();

	hipMemcpy(matDst, param->devMatDst, width * height * sizeof(int), hipMemcpyDeviceToHost);
}

#else

/* copy from host to device only at the first time */
void logicForOneGeneration(ALGORITHM_CUDA_NORMAL_PARAM *param, int* matDst, int* matSrc, int width, int height)
{
	if (param->isFirstOperation != 0) {
		/* after the 2nd time, devMatSrc is copied from devMatDst */
		hipMemcpy(param->devMatSrc, matSrc, width * height * sizeof(int), hipMemcpyHostToDevice);
		param->isFirstOperation = 0;
	}

	int blocksizeW = 16;
	int blocksizeH = 16;
	dim3 block(blocksizeW, blocksizeH);
	dim3 grid(width / blocksizeW, height / blocksizeH);
	loop << <grid, block >> > (param->devMatDst, param->devMatSrc, width, height);
	hipDeviceSynchronize();
	hipMemcpy(param->devMatSrc, param->devMatDst, width * height * sizeof(int), hipMemcpyDeviceToDevice);

	hipMemcpy(matDst, param->devMatDst, width * height * sizeof(int), hipMemcpyDeviceToHost);
}
#endif


void cudaInitialize(ALGORITHM_CUDA_NORMAL_PARAM *param, int width, int height)
{
	hipMalloc((void**)&param->devMatSrc, width * height * sizeof(int));
	hipMalloc((void**)&param->devMatDst, width * height * sizeof(int));
	param->isFirstOperation = 1;
}

void cudaFinalize(ALGORITHM_CUDA_NORMAL_PARAM *param)
{
	hipFree(param->devMatSrc);
	hipFree(param->devMatDst);
	hipDeviceReset();
}


/*
 * Don't use hipMallocManaged
 * Memory access exception occurs when I call logicForOneGeneration from several threads
 */
#if 0
void allocManaged(int **p, int size)
{
	hipMallocManaged(p, size);
}

void freeManaged(int *p)
{
	hipFree(p);
}

void cudaDeviceSynchronizeWrapper()
{
	hipDeviceSynchronize();
}

void logicForOneGeneration(int* matDst, int* matSrc, int width, int height)
{
	int blocksizeW = 16;
	int blocksizeH = 16;
	dim3 block(blocksizeW, blocksizeH);
	dim3 grid(width / blocksizeW, height / blocksizeH);
	loop << <grid, block >> > (matDst, matSrc, width, height);
	hipDeviceSynchronize();
}
#endif



}

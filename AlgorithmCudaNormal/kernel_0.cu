#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "stdlib.h"
#include <string.h>
#include "algorithmCudaNormal.h"
#include "algorithmCudaNormalInternal.h"

namespace AlgorithmCudaNormal
{
#if 0
}	// indent guard
#endif

__forceinline__ __device__ void updateCell(int* matDst, int* matSrc, int globalIndex, int cnt)
{
	if (matSrc[globalIndex] == 0) {
		if (cnt == 3) {
			// birth
			matDst[globalIndex] = 1;
		} else {
			// keep dead
			matDst[globalIndex] = 0;
		}
	} else {
		if (cnt <= 2 || cnt >= 5) {
			// die
			matDst[globalIndex] = 0;
		} else {
			// keep alive (age++)
			matDst[globalIndex] = matSrc[globalIndex] + 1;
		}
	}
}

__global__ void loop_0(int* matDst, int *matSrc, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	//if (x >= width || y >= height) {
	//	printf("%d %d\n", x, y);
	//	return;
	//}

	register int cnt = 0;
	for (int yy = y - 1; yy <= y + 1; yy++) {
		int roundY = yy;
		if (roundY >= height) roundY = 0;
		if (roundY < 0) roundY = height - 1;
		for (int xx = x - 1; xx <= x + 1; xx++) {
			int roundX = xx;
			if (roundX >= width) roundX = 0;
			if (roundX < 0) roundX = width - 1;
			if (matSrc[width * roundY + roundX] != 0) {
				cnt++;
			}
		}
	}
	updateCell(matDst, matSrc, y * width + x, cnt);
}

/* The most basic algorithm
 */
void process_0(ALGORITHM_CUDA_NORMAL_PARAM *param, int width, int height)
{
	dim3 block(BLOCK_SIZE_W, BLOCK_SIZE_H);
	dim3 grid(width / BLOCK_SIZE_W, height / BLOCK_SIZE_H);

	CHECK(hipMemcpy(param->devMatSrc, param->hostMatSrc, width * height * sizeof(int), hipMemcpyHostToDevice));

	loop_0 <<< grid, block >>> (param->devMatDst, param->devMatSrc, width, height);
	CHECK(hipDeviceSynchronize());

	CHECK(hipMemcpy(param->hostMatDst, param->devMatDst, width * height * sizeof(int), hipMemcpyDeviceToHost));

	swapMat(param);

	// hostMatSrc is ready to be displayed
}


}
